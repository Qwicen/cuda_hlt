#include "hip/hip_runtime.h"
#include "RawBankDecoder.cuh"
#include "assert.h"

using namespace SciFi;

__device__ uint32_t channelInBank(uint32_t c) {
  return (c >> SciFiRawBankParams::cellShift);
}

__device__ uint16_t getLinkInBank(uint16_t c){
  return (c >> SciFiRawBankParams::linkShift);
}

__device__ int cell(uint16_t c) {
  return (c >> SciFiRawBankParams::cellShift     ) & SciFiRawBankParams::cellMaximum;
}

__device__ int fraction(uint16_t c) {
  return (c >> SciFiRawBankParams::fractionShift ) & SciFiRawBankParams::fractionMaximum;
}

__device__ bool cSize(uint16_t c) {
  return (c >> SciFiRawBankParams::sizeShift     ) & SciFiRawBankParams::sizeMaximum;
}

__global__ void raw_bank_decoder(
  char *scifi_events,
  uint *scifi_event_offsets,
  uint *scifi_hit_count,
  uint *scifi_hits,
  char *scifi_geometry
) {
  // maybe not hardcoded, or in another place
  const float invClusRes[] = {1/0.05, 1/0.08, 1/0.11, 1/0.14, 1/0.17, 1/0.20, 1/0.23, 1/0.26, 1/0.29};
  const uint32_t number_of_events = gridDim.x;
  const uint32_t event_number = blockIdx.x;

  SciFiGeometry geom(scifi_geometry);
  const auto event = SciFiRawEvent(scifi_events + scifi_event_offsets[event_number]);

  SciFiHits hits;
  hits.typecast_unsorted(scifi_hits, scifi_hit_count[number_of_events * SciFi::Constants::n_zones]);
  SciFiHitCount hit_count;
  hit_count.typecast_after_prefix_sum(scifi_hit_count, event_number, number_of_events);

  __shared__ uint32_t shared_layer_offsets[SciFi::Constants::n_zones];

  for (uint i = threadIdx.x; i < SciFi::Constants::n_zones; i += blockDim.x) {
    shared_layer_offsets[i] = hit_count.layer_offsets[i];
  }

  for (uint i = threadIdx.x; i < SciFi::Constants::n_zones; i += blockDim.x) {
    hit_count.n_hits_layers[i] = 0;
  }

  __syncthreads();

  // Merge of PrStoreFTHit and RawBankDecoder.
  auto make_cluster = [&](uint32_t chan, uint8_t fraction, uint8_t pseudoSize) {
    const SciFi::SciFiChannelID id(chan);

    // Offset to save space in geometry structure, see DumpFTGeometry.cpp
    const uint32_t mat = id.uniqueMat() - 512;
    const uint32_t iQuarter = id.uniqueQuarter() - 16;
    const uint32_t planeCode = id.uniqueLayer() - 4;
    // See Kernel/LHCbID.h. Maybe no hardcoding?
    const uint32_t lhcbid = (10u << 28) + chan;
    const float dxdy = geom.dxdy[mat];
    const float dzdy = geom.dzdy[mat];
    const float globaldy = geom.globaldy[mat];
    float uFromChannel = geom.uBegin[mat] + (2 * id.channel() + 1 + fraction) * geom.halfChannelPitch[mat];
    if( id.die() ) uFromChannel += geom.dieGap[mat];
    uFromChannel += id.sipm() * geom.sipmPitch[mat];
    const float endPointX = geom.mirrorPointX[mat] + geom.ddxX[mat] * uFromChannel;
    const float endPointY = geom.mirrorPointY[mat] + geom.ddxY[mat] * uFromChannel;
    const float endPointZ = geom.mirrorPointZ[mat] + geom.ddxZ[mat] * uFromChannel;
    const float x0 = endPointX - dxdy * endPointY;
    const float z0 = endPointZ - dzdy * endPointY;

    // ORIGINAL: if(id.isBottom()) std::swap(yMin, yMax);
    float yMin = endPointY + id.isBottom() * globaldy;
    float yMax = endPointY + !id.isBottom() * globaldy;

    assert( pseudoSize < 9 && "Pseudosize of cluster is > 8. Out of range.");
    float werrX = invClusRes[pseudoSize];

    // Apparently the unique* methods are not designed to start at 0, therefore -16
    const uint32_t uniqueZone = ((id.uniqueQuarter() - 16) >> 1);
    uint32_t* hits_zone = hit_count.n_hits_layers + uniqueZone;
    uint32_t hitIndex = atomicAdd(hits_zone, 1);

    assert( hitIndex < hit_count.n_hits_layers[uniqueZone] );
    
    hitIndex += shared_layer_offsets[uniqueZone];

    hits.x0[hitIndex] = x0;
    hits.z0[hitIndex] = z0;
    hits.w[hitIndex] = werrX * werrX;
    hits.dxdy[hitIndex] = dxdy;
    hits.dzdy[hitIndex] = dzdy;
    hits.yMin[hitIndex] = yMin;
    hits.yMax[hitIndex] = yMax;
    hits.LHCbID[hitIndex] = lhcbid;
    hits.planeCode[hitIndex] = 2 * planeCode + (uniqueZone % 2); //  planeCode;
    hits.hitZone[hitIndex] = uniqueZone % 2;
  };

  // copied straight from FTRawBankDecoder.cpp
  auto make_clusters = [&](uint32_t firstChannel, uint16_t c, uint16_t c2) {
    unsigned int delta = (cell(c2) - cell(c));

    // fragmented clusters, size > 2*max size
    // only edges were saved, add middles now
    if ( delta  > SciFiRawBankParams::clusterMaxWidth ) {
      //add the first edge cluster, and then the middle clusters
      for(unsigned int  i = SciFiRawBankParams::clusterMaxWidth; i < delta ; i+= SciFiRawBankParams::clusterMaxWidth){
        // all middle clusters will have same size as the first cluster,
        // so re-use the fraction
        make_cluster( firstChannel+i, fraction(c), 0 );
      }
      //add the last edge
      make_cluster  ( firstChannel+delta, fraction(c2), 0 );
    } else { //big cluster size upto size 8
      unsigned int widthClus  =  2 * delta - 1 + fraction(c2);
      make_cluster( firstChannel+(widthClus-1)/2 - int((SciFiRawBankParams::clusterMaxWidth - 1)/2),
                    (widthClus-1)%2, widthClus );
    }//end if adjacent clusters
  };//End lambda make_clusters

  // Main execution loop
  for(uint i = threadIdx.x; i < event.number_of_raw_banks; i += blockDim.x)
  {
    auto rawbank = event.getSciFiRawBank(i);
    uint16_t* it = rawbank.data + 2;
    uint16_t* last = rawbank.last;
    if (*(last-1) == 0) --last;//Remove padding at the end
    for( ;  it < last; ++it ){ // loop over the clusters
      uint16_t c = *it;
      uint32_t ch = geom.bank_first_channel[rawbank.sourceID] + channelInBank(c);

      if( !cSize(c) || it+1 == last ) { //No size flag or last cluster
        make_cluster(ch, fraction(c), 4);
      } else {//Flagged or not the last one.
        unsigned c2 = *(it+1);
        if( cSize(c2) && getLinkInBank(c) == getLinkInBank(c2) ) {
          make_clusters(ch,c,c2);
          ++it;
        } else {
          make_cluster(ch, fraction(c), 4);
        }
      }
    }
  }
}
