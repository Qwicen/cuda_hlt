#include "hip/hip_runtime.h"
#include "PrForwardTools.cuh"

/* Look first in x layers, then in stereo layers for hits
   do 1D Hough transform for x- and stereo hits
   do global 1D Hough transform
   use TMVAs to obtain track quality */
__host__ __device__ void find_forward_tracks(
  const SciFi::SciFiHits& scifi_hits,
  const SciFi::SciFiHitCount& scifi_hit_count,
  const VeloUTTracking::TrackUT& veloUTTrack,
  SciFi::Track* outputTracks,
  uint* n_forward_tracks,
  SciFi::Tracking::TMVA* tmva1,
  SciFi::Tracking::TMVA* tmva2,
  SciFi::Tracking::Arrays* constArrays,
  const MiniState& velo_state
) {

  // The LHCb framework code had a PT preselection for the VeloUT tracks
  // here, which I am removing because this should be done explicitly through
  // track selectors if we do it at all, not hacked inside the tracking code

  // Some values related to the forward track which were stored in a dedicated
  // forward track class, let's see if I can get rid of that here
  const float zRef_track    = SciFi::Tracking::zReference;
  const float xAtRef = xFromVelo( zRef_track, velo_state );
  const float xParams_seed[4] = {xAtRef, velo_state.tx, 0.f, 0.f};
  const float yAtRef = yFromVelo( zRef_track, velo_state );
  const float yParams_seed[4] = {yAtRef, velo_state.ty, 0.f, 0.f};

  // First loop Hough cluster search, set initial search windows
  SciFi::Tracking::HitSearchCuts pars_first{SciFi::Tracking::minXHits, SciFi::Tracking::maxXWindow, SciFi::Tracking::maxXWindowSlope, SciFi::Tracking::maxXGap, 4u};
  SciFi::Tracking::HitSearchCuts pars_second{SciFi::Tracking::minXHits_2nd, SciFi::Tracking::maxXWindow_2nd, SciFi::Tracking::maxXWindowSlope_2nd, SciFi::Tracking::maxXGap_2nd, 4u};

  int allXHits[2][SciFi::Tracking::max_x_hits];
  int n_x_hits[2] = {0};
  float coordX[2][SciFi::Tracking::max_x_hits];
  
  if(yAtRef>-5.f)
    collectAllXHits(
      scifi_hits, scifi_hit_count, allXHits[1], n_x_hits[1],
      coordX[1], xParams_seed, yParams_seed, constArrays,
      velo_state, veloUTTrack.qop, 1); 
  if(yAtRef< 5.f)
    collectAllXHits(
      scifi_hits, scifi_hit_count, allXHits[0], n_x_hits[0],
      coordX[0], xParams_seed, yParams_seed, constArrays,
      velo_state, veloUTTrack.qop, -1);

  SciFi::Tracking::Track candidate_tracks[SciFi::Tracking::max_candidate_tracks];
  int n_candidate_tracks = 0;
  bool usedHits[2][SciFi::Tracking::max_x_hits] = {false};
  
  if(yAtRef>-5.f)selectXCandidates(
    scifi_hits, scifi_hit_count, allXHits[1], n_x_hits[1],
    usedHits[1], coordX[1], veloUTTrack,
    candidate_tracks, n_candidate_tracks,
    zRef_track, xParams_seed, yParams_seed,
    velo_state, pars_first,  constArrays, 1, false);
  if(yAtRef< 5.f)selectXCandidates(
    scifi_hits, scifi_hit_count, allXHits[0], n_x_hits[0],
    usedHits[0], coordX[0], veloUTTrack,
    candidate_tracks, n_candidate_tracks,
    zRef_track, xParams_seed, yParams_seed,
    velo_state, pars_first, constArrays, -1, false); 
  
  SciFi::Tracking::Track selected_tracks[SciFi::Tracking::max_selected_tracks];
  int n_selected_tracks = 0;
    
  selectFullCandidates(
    scifi_hits, scifi_hit_count,
    candidate_tracks,
    n_candidate_tracks,
    selected_tracks,
    n_selected_tracks,
    xParams_seed, yParams_seed,
    velo_state, veloUTTrack.qop,
    pars_first, tmva1, tmva2, constArrays, false);
 
  bool ok = false;
  for ( int i_track = 0; i_track < n_selected_tracks; ++i_track ) {
    if ( selected_tracks[i_track].hitsNum > 10 )
      ok = true;
  }
  assert( n_selected_tracks < SciFi::Tracking::max_selected_tracks );

  SciFi::Tracking::Track candidate_tracks2[SciFi::Tracking::max_tracks_second_loop];
  int n_candidate_tracks2 = 0;

  if (!ok && SciFi::Tracking::secondLoop) { // If you found nothing begin the 2nd loop
    if(yAtRef>-5.f)selectXCandidates(
      scifi_hits, scifi_hit_count, allXHits[1], n_x_hits[1],
      usedHits[1], coordX[1], veloUTTrack,
      candidate_tracks2, n_candidate_tracks2,
      zRef_track, xParams_seed, yParams_seed,
      velo_state, pars_second, constArrays, 1, true);
    if(yAtRef< 5.f)selectXCandidates(
      scifi_hits, scifi_hit_count, allXHits[0], n_x_hits[0],
      usedHits[0], coordX[0], veloUTTrack,
      candidate_tracks2, n_candidate_tracks2,
      zRef_track, xParams_seed, yParams_seed,
      velo_state, pars_second, constArrays, -1, true);  

    SciFi::Tracking::Track selected_tracks2[SciFi::Tracking::max_tracks_second_loop];
    int n_selected_tracks2 = 0;
    
    selectFullCandidates(
      scifi_hits, scifi_hit_count,
      candidate_tracks2,
      n_candidate_tracks2,
      selected_tracks2,
      n_selected_tracks2,
      xParams_seed, yParams_seed,
      velo_state, veloUTTrack.qop,
      pars_second, tmva1, tmva2, constArrays, true);
 
    for ( int i_track = 0; i_track < n_selected_tracks2; ++i_track ) {
      assert( n_selected_tracks < SciFi::Tracking::max_selected_tracks );
      selected_tracks[n_selected_tracks++] = selected_tracks2[i_track];
    }
   
    ok = (n_selected_tracks > 0);
  }
 
  if(ok || !SciFi::Tracking::secondLoop){

    if ( n_selected_tracks > 1 ) {
      // not using thrust::sort due to temporary_buffer::allocate:: get_temporary_buffer failed" error
      //thrust::sort( thrust::seq, selected_tracks, selected_tracks + n_selected_tracks, lowerByQuality);
      sort_tracks( 
        selected_tracks, 
        n_selected_tracks,
        [] (SciFi::Tracking::Track t1, SciFi::Tracking::Track t2) {
          if ( t1.quality < t2.quality ) return -1;
          if ( t1.quality == t2.quality ) return 0;
          return 1;
        }
      );
      
    }

    float minQuality = SciFi::Tracking::maxQuality;
    for ( int i_track = 0; i_track < n_selected_tracks; ++i_track ) {
      SciFi::Tracking::Track& track = selected_tracks[i_track];
      if(track.quality + SciFi::Tracking::deltaQuality < minQuality)
        minQuality = track.quality + SciFi::Tracking::deltaQuality;
      if(!(track.quality > minQuality)) {
        
        SciFi::Track tr = makeTrack( track );
        // add LHCbIDs from Velo and UT part of the track
        for ( int i_hit = 0; i_hit < veloUTTrack.hitsNum; ++i_hit ) {
          tr.addLHCbID( veloUTTrack.LHCbIDs[i_hit] );
        }
        if ( tr.hitsNum >= VeloUTTracking::max_track_size )
          printf("veloUT track hits Num = %u \n", tr.hitsNum );
        assert( tr.hitsNum < VeloUTTracking::max_track_size );
        
        // add LHCbIDs from SciFi part of the track
        for ( int i_hit = 0; i_hit < track.hitsNum; ++i_hit ) {
          tr.addLHCbID( scifi_hits.LHCbID[ track.hit_indices[i_hit] ] );
        }
        assert( tr.hitsNum < SciFi::max_track_size );

        if ( *n_forward_tracks >= SciFi::max_tracks )
          printf("n_forward_tracks = %u \n", *n_forward_tracks);
        assert(*n_forward_tracks < SciFi::max_tracks );
#ifndef __CUDA_ARCH__
        outputTracks[(*n_forward_tracks)++] = tr;
#else
        uint n_tracks = atomicAdd(n_forward_tracks, 1);
        assert( n_tracks < SciFi::max_tracks );
        outputTracks[n_tracks] = tr;
#endif
      }
    }
  }  
    
}

// Turn SciFi::Tracking::Track into a SciFi::Track
__host__ __device__ SciFi::Track makeTrack( SciFi::Tracking::Track track ) {
  SciFi::Track tr;
  tr.qop     = track.qop;
  tr.chi2    = track.chi2;

  return tr;
}

//=========================================================================
//  Create Full candidates out of xCandidates
//  Searching for stereo hits
//  Fit of all hits
//  save everything in track candidate folder
//=========================================================================
__host__ __device__ void selectFullCandidates(
  const SciFi::SciFiHits& scifi_hits,
  const SciFi::SciFiHitCount& scifi_hit_count,
  SciFi::Tracking::Track* candidate_tracks,
  int& n_candidate_tracks,
  SciFi::Tracking::Track* selected_tracks,
  int& n_selected_tracks,
  const float xParams_seed[4],
  const float yParams_seed[4],
  MiniState velo_state,
  const float VeloUT_qOverP,
  SciFi::Tracking::HitSearchCuts& pars,
  SciFi::Tracking::TMVA* tmva1,
  SciFi::Tracking::TMVA* tmva2,
  SciFi::Tracking::Arrays* constArrays,
  bool secondLoop)
{

  PlaneCounter planeCounter;
  planeCounter.clear();
  float mlpInput[7] = {0};
  
  for ( int i_track = 0; i_track < n_candidate_tracks; ++i_track ) {
    SciFi::Tracking::Track* cand = candidate_tracks + i_track;
    
    pars.minStereoHits = 4;

    if(cand->hitsNum + pars.minStereoHits < SciFi::Tracking::minTotalHits) {
      pars.minStereoHits = SciFi::Tracking::minTotalHits - cand->hitsNum;
    }
    
    int stereoHits[SciFi::Tracking::max_stereo_hits];
    int n_stereoHits = 0;
    float stereoCoords[SciFi::Tracking::max_stereo_hits];
    collectStereoHits(
      scifi_hits, scifi_hit_count,
      *cand, velo_state,
      pars, constArrays, stereoCoords, 
      stereoHits, n_stereoHits);

    if(n_stereoHits < pars.minStereoHits) continue;
    
    if ( !selectStereoHits(
      scifi_hits, scifi_hit_count,
      *cand, constArrays,
      stereoCoords, stereoHits, n_stereoHits,
      velo_state, pars) ) continue;

    planeCounter.clear();
    for ( int i_hit = 0; i_hit < cand->hitsNum; ++i_hit ) {
      int hit = cand->hit_indices[i_hit];
      planeCounter.addHit( scifi_hits.planeCode[hit]/2 );
    }
    
    //make a fit of ALL hits using their x coordinate
    if(!quadraticFitX(scifi_hits, cand->trackParams, cand->hit_indices, cand->hitsNum, planeCounter, pars))continue;
 
    //track has enough hits, calcualte quality and save if good enough
    if(planeCounter.nbDifferent >= SciFi::Tracking::minTotalHits){

      const float qOverP  = calcqOverP(cand->trackParams[1], constArrays, velo_state);
      //orig params before fitting , TODO faster if only calc once?? mem usage?
      const float xAtRef = cand->trackParams[0];
      float dSlope  = ( velo_state.x + (SciFi::Tracking::zReference - velo_state.z) * velo_state.tx - xAtRef ) / ( SciFi::Tracking::zReference - constArrays->zMagnetParams[0]);
      const float zMagSlope = constArrays->zMagnetParams[2] * pow(velo_state.tx,2) +  constArrays->zMagnetParams[3] * pow(velo_state.ty,2);
      const float zMag    = constArrays->zMagnetParams[0] + constArrays->zMagnetParams[1] *  dSlope * dSlope  + zMagSlope;
      const float xMag    = velo_state.x + (zMag- velo_state.z) * velo_state.tx;
      const float slopeT  = ( xAtRef - xMag ) / ( SciFi::Tracking::zReference - zMag );
      dSlope        = slopeT - velo_state.tx;
      const float dyCoef  = dSlope * dSlope * velo_state.ty;

      float bx = slopeT;
      float ay = velo_state.y + (SciFi::Tracking::zReference - velo_state.z) * velo_state.ty;
      float by = velo_state.ty + dyCoef * SciFi::Tracking::byParams;

      //ay,by,bx params
      const float ay1  = cand->trackParams[4];
      const float by1  = cand->trackParams[5];
      const float bx1  = cand->trackParams[1];

      mlpInput[0] = planeCounter.nbDifferent;
      mlpInput[1] = qOverP;
      mlpInput[2] = VeloUT_qOverP - qOverP; //veloUT - scifi
      if(fabsf(VeloUT_qOverP) < 1e-9f) mlpInput[2] = 0.f; //no momentum estiamte
      mlpInput[3] = pow(velo_state.tx,2) + pow(velo_state.ty,2);
      mlpInput[4] = by - by1;
      mlpInput[5] = bx - bx1;
      mlpInput[6] = ay - ay1;

      float quality = 0.f;
      /// WARNING: if the NN classes straight out of TMVA are used, put a mutex here!
      if(pars.minXHits > 4) quality = GetMvaValue(mlpInput, tmva1); //1st loop NN
      else                  quality = GetMvaValue(mlpInput, tmva2); //2nd loop NN

      quality = 1.f-quality; //backward compability

      if(quality < SciFi::Tracking::maxQuality){
        cand->quality = quality;
        cand->set_qop( qOverP );
        if (!secondLoop) 
          assert (n_selected_tracks < SciFi::Tracking::max_selected_tracks );
        else if (secondLoop)
          assert (n_selected_tracks < SciFi::Tracking::max_tracks_second_loop );
        selected_tracks[n_selected_tracks++] = *cand;
        if (!secondLoop) {
          if ( n_selected_tracks >= SciFi::Tracking::max_selected_tracks ) break;
        }
        else if ( secondLoop ) {
          if ( n_selected_tracks >= SciFi::Tracking::max_tracks_second_loop ) break;
        }
          
      }  
    }
  }
}
