#include "hip/hip_runtime.h"
#include "VeloUT.cuh"

__global__ void veloUT(
  uint* dev_ut_hits,
  uint* dev_ut_hit_offsets,
  int* dev_atomics_storage,
  uint* dev_velo_track_hit_number,
  uint* dev_velo_track_hits,
  uint* dev_velo_states,
  VeloUTTracking::TrackUT* dev_veloUT_tracks,
  int* dev_atomics_veloUT,
  PrUTMagnetTool* dev_ut_magnet_tool,
  float* dev_ut_dxDy,
  const uint* dev_unique_x_sector_layer_offsets,
  const uint* dev_unique_x_sector_offsets,
  const float* dev_unique_sector_xs
) {
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const uint number_of_unique_x_sectors = dev_unique_x_sector_layer_offsets[4];
  const uint total_number_of_hits = dev_ut_hit_offsets[number_of_events * number_of_unique_x_sectors];
  
  // Velo consolidated types
  const Velo::Consolidated::Tracks velo_tracks {(uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {dev_velo_states, velo_tracks.total_number_of_tracks};
  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  UTHitOffsets ut_hit_offsets {dev_ut_hit_offsets, event_number, number_of_unique_x_sectors, dev_unique_x_sector_layer_offsets};
  UTHits ut_hits {dev_ut_hits, total_number_of_hits};
  
  /* dev_atomics_veloUT contains in an SoA:
     1. # of veloUT tracks
     2. # velo tracks in UT acceptance
  */
  int* n_veloUT_tracks_event = dev_atomics_veloUT + event_number;
  VeloUTTracking::TrackUT* veloUT_tracks_event = dev_veloUT_tracks + event_number * VeloUTTracking::max_num_tracks;
  int* n_velo_tracks_in_UT_event = dev_atomics_veloUT + number_of_events + event_number;
  
  // initialize atomic veloUT tracks counter
  if ( threadIdx.x == 0 ) {
    *n_veloUT_tracks_event = 0;
    *n_velo_tracks_in_UT_event = 0;
  }
  __syncthreads();

  // if (threadIdx.x == 0) {
  //   for (int i=0; i<4; ++i) {
  //     printf("Layer %i hits:\n", i);

  //     for (int s=dev_unique_x_sector_layer_offsets[i]; s<dev_unique_x_sector_layer_offsets[i+1]; ++s) {
  //       printf(" Sector group %i, x %f:\n", s, dev_unique_sector_xs[s]);
  //       uint group_offset = ut_hit_offsets.sector_group_offset(s);
  //       uint n_hits_group = ut_hit_offsets.sector_group_number_of_hits(s);

  //       for (int j=0; j<n_hits_group; ++j) {
  //         const auto hit_index = group_offset + j;

  //         printf("  yBegin = %f, yEnd = %f, zAtYEq0 = %f, xAtYEq0 = %f, weight = %f, highThreshold = %u \n",
  //          ut_hits.yBegin[hit_index],
  //          ut_hits.yEnd[hit_index],
  //          ut_hits.zAtYEq0[hit_index],
  //          ut_hits.xAtYEq0[hit_index],
  //          ut_hits.weight[hit_index],
  //          ut_hits.highThreshold[hit_index]);
  //       }
  //     }
  //   }
  // }

  const float* fudgeFactors = &(dev_ut_magnet_tool->dxLayTable[0]);
  const float* bdlTable     = &(dev_ut_magnet_tool->bdlTable[0]);

  // array to store indices of selected hits in layers
  // -> can then access the hit information in the HitsSoA
  int hitCandidatesInLayers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer];
  int n_hitCandidatesInLayers[VeloUTTracking::n_layers];
  
  for ( int i = 0; i < (number_of_tracks_event + blockDim.x - 1) / blockDim.x; ++i) {
    const int i_track = i * blockDim.x + threadIdx.x;
    
    const uint velo_states_index = event_tracks_offset + i_track;
    if (i_track >= number_of_tracks_event) continue;
    if (velo_states.backward[velo_states_index]) continue;

    // Mini State with only x, y, tx, ty and z
    MiniState velo_state {velo_states, velo_states_index};

    if(!veloTrackInUTAcceptance(velo_state)) continue;

    atomicAdd(n_velo_tracks_in_UT_event, 1);

     // for storing calculated x position of hits for this track
    float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer];

    for ( int i_layer = 0; i_layer < VeloUTTracking::n_layers; ++i_layer ) {
      n_hitCandidatesInLayers[i_layer] = 0;
    }

    if( !getHits(
          hitCandidatesInLayers,
          n_hitCandidatesInLayers,
          x_pos_layers,
          ut_hits,
          ut_hit_offsets,
          fudgeFactors,
          velo_state,
          dev_ut_dxDy,
          dev_unique_sector_xs,
          dev_unique_x_sector_layer_offsets)
        ) continue;

    TrackHelper helper {velo_state};

    // indices within hitCandidatesInLayers for selected hits belonging to best track 
    int hitCandidateIndices[VeloUTTracking::n_layers];
    
    // go through UT layers in forward direction
    if(!formClusters(
          hitCandidatesInLayers,
          n_hitCandidatesInLayers,
          x_pos_layers,
          hitCandidateIndices,
          ut_hits,
          ut_hit_offsets,
          helper,
          velo_state,
          dev_ut_dxDy,
          true)) {
      
      // go through UT layers in backward direction
      formClusters(
        hitCandidatesInLayers,
        n_hitCandidatesInLayers,
        x_pos_layers,
        hitCandidateIndices,
        ut_hits,
        ut_hit_offsets,
        helper,
        velo_state,
        dev_ut_dxDy,
        false);
    }
    
    if ( helper.n_hits > 0 ) {
      const uint velo_track_hit_number = velo_tracks.number_of_hits(i_track);
      const Velo::Consolidated::Hits velo_track_hits = velo_tracks.get_hits(dev_velo_track_hits, i_track);

      prepareOutputTrack(
        velo_track_hits,
        velo_track_hit_number,
        helper,
        velo_state,
        hitCandidatesInLayers,
        n_hitCandidatesInLayers,
        ut_hits,
        ut_hit_offsets,
        x_pos_layers,
        hitCandidateIndices,
        veloUT_tracks_event,
        n_veloUT_tracks_event,
        i_track,
        bdlTable);
    }
    
  } // velo tracks
 
}
