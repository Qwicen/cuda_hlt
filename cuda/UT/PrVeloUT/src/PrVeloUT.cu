#include "hip/hip_runtime.h"
#include "PrVeloUT.cuh"

//-----------------------------------------------------------------------------
// Implementation file for PrVeloUT
//
// 2007-05-08: Mariusz Witek
// 2017-03-01: Christoph Hasse (adapt to future framework)
// 2018-05-05: Plácido Fernández (make standalone)
// 2018-07:    Dorothea vom Bruch (convert to C and then CUDA code)
//-----------------------------------------------------------------------------

// -- These things are all hardcopied from the PrTableForFunction
// -- and PrUTMagnetTool
// -- If the granularity or whatever changes, this will give wrong results

  __host__ __device__ int masterIndex(const int index1, const int index2, const int index3){
    return (index3*11 + index2)*31 + index1;
  }


//=====================================================================
// Propagate to end of Velo z position (z=770mm)
// only propagate x, y, z; covariance matrix is not needed
//=====================================================================
__host__ __device__ void propagate_state_to_end_velo(
  Velo::State& velo_state
) {
  const float dz = VeloTracking::z_endVelo - velo_state.z;
  velo_state.x += dz * velo_state.tx;
  velo_state.y += dz * velo_state.ty;
  velo_state.z = VeloTracking::z_endVelo;
}


//=============================================================================
// Reject tracks outside of acceptance or pointing to the beam pipe
//=============================================================================
__host__ __device__ bool veloTrackInUTAcceptance(
  const MiniState& state
) {
  const float xMidUT = state.x + state.tx*( PrVeloUTConst::zMidUT - state.z);
  const float yMidUT = state.y + state.ty*( PrVeloUTConst::zMidUT - state.z);

  if( xMidUT*xMidUT+yMidUT*yMidUT  < PrVeloUTConst::centralHoleSize*PrVeloUTConst::centralHoleSize ) return false;
  if( (std::abs(state.tx) > PrVeloUTConst::maxXSlope) || (std::abs(state.ty) > PrVeloUTConst::maxYSlope) ) return false;

  if(PrVeloUTConst::passTracks && std::abs(xMidUT) < PrVeloUTConst::passHoleSize && std::abs(yMidUT) < PrVeloUTConst::passHoleSize) {
    return false;
  }

  return true;
}

//=============================================================================
// Find the hits
//=============================================================================
__device__ bool getHits(
  int hitCandidatesInLayers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  int n_hitCandidatesInLayers[VeloUTTracking::n_layers],
  float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  UTHits& ut_hits,
  UTHitOffsets& ut_hit_offsets,
  const float* fudgeFactors, 
  const MiniState& trState,
  const float* ut_dxDy,
  const float* dev_unique_sector_xs,
  const uint* dev_unique_x_sector_layer_offsets)
{
  // -- This is hardcoded, so faster
  // -- If you ever change the Table in the magnet tool, this will be wrong
  const float absSlopeY = std::abs( trState.ty );
  const int index = (int)(absSlopeY*100 + 0.5);
  assert( 3 + 4*index < PrUTMagnetTool::N_dxLay_vals );
  const std::array<float,4> normFact = { 
    fudgeFactors[4*index], 
    fudgeFactors[1 + 4*index], 
    fudgeFactors[2 + 4*index], 
    fudgeFactors[3 + 4*index] 
  };

  // -- this 500 seems a little odd...
  // to do: change back!
  const float invTheta = std::min(500., 1.0/std::sqrt(trState.tx*trState.tx+trState.ty*trState.ty));
  //const float minMom   = std::max(PrVeloUTConst::minPT*invTheta, PrVeloUTConst::minMomentum);
  const float minMom   = std::max(PrVeloUTConst::minPT*invTheta, float(1.5)*Gaudi::Units::GeV);
  const float xTol     = std::abs(1. / ( PrVeloUTConst::distToMomentum * minMom ));
  const float yTol     = PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * xTol;

  int nLayers = 0;

  float dxDyHelper[VeloUTTracking::n_layers] = {0., 1., -1., 0};
  for(int iStation = 0; iStation < 2; ++iStation) {

    if( iStation == 1 && nLayers == 0 ) return false;

    for(int iLayer = 0; iLayer < 2; ++iLayer) {
      if( iStation == 1 && iLayer == 1 && nLayers < 2 ) return false;

      int layer = 2*iStation+iLayer;
      int layer_offset = ut_hit_offsets.layer_offset(layer);
      
      if( ut_hit_offsets.layer_number_of_hits(layer) == 0 ) continue;
      const float dxDy   = ut_dxDy[layer];
      const float zLayer = ut_hits.zAtYEq0[layer_offset + 0]; 

      const float yAtZ   = trState.y + trState.ty*(zLayer - trState.z);
      const float xLayer = trState.x + trState.tx*(zLayer - trState.z);
      const float yLayer = yAtZ + yTol * dxDyHelper[layer];

      const float normFactNum = normFact[2*iStation + iLayer];
      const float invNormFact = 1.0/normFactNum;

      const float lowerBoundX =
        (xLayer - dxDy*yLayer) - xTol*invNormFact - std::abs(trState.tx)*PrVeloUTConst::intraLayerDist;
      const float upperBoundX =
        (xLayer - dxDy*yLayer) + xTol*invNormFact + std::abs(trState.tx)*PrVeloUTConst::intraLayerDist;

      // Find sector group for lowerBoundX and upperBoundX
      const uint first_sector_group_in_layer = dev_unique_x_sector_layer_offsets[layer];
      const uint last_sector_group_in_layer = dev_unique_x_sector_layer_offsets[layer+1];
      const uint number_of_sector_groups = last_sector_group_in_layer - first_sector_group_in_layer;

      uint lowerBoundSectorGroup = first_sector_group_in_layer;
      uint upperBoundSectorGroup = last_sector_group_in_layer - 1;

      // The window of search is out of bounds
      if (upperBoundX < dev_unique_sector_xs[first_sector_group_in_layer] ||
          lowerBoundX > dev_unique_sector_xs[last_sector_group_in_layer - 1]) {
        continue;
      }

      for (int i=first_sector_group_in_layer + 2; i<last_sector_group_in_layer; ++i) {
        if (dev_unique_sector_xs[i] > lowerBoundX) {
          lowerBoundSectorGroup = i-2;
          break;
        }
      }

      for (int i=0; i<number_of_sector_groups; ++i) {
        const uint current_sector_group = first_sector_group_in_layer + i;
        if (dev_unique_sector_xs[current_sector_group] > upperBoundX) {
          if (i == number_of_sector_groups - 1) {
            upperBoundSectorGroup = current_sector_group;
          } else {
            upperBoundSectorGroup = current_sector_group + 1;
          }
          break;
        }
      }

      assert(upperBoundSectorGroup < last_sector_group_in_layer);
      assert(lowerBoundSectorGroup >= first_sector_group_in_layer);
      assert(lowerBoundSectorGroup < upperBoundSectorGroup);

      findHits(lowerBoundSectorGroup, upperBoundSectorGroup,
        ut_hits, ut_hit_offsets, layer_offset, layer, ut_dxDy,
        trState, xTol*invNormFact, invNormFact,
        hitCandidatesInLayers[layer], n_hitCandidatesInLayers[layer],
        x_pos_layers);

      nLayers += int( !( n_hitCandidatesInLayers[layer] == 0 ) );
    }
  }

  return nLayers > 2;
}

//=========================================================================
// Form clusters
//=========================================================================
__host__ __device__ bool formClusters(
  const int hitCandidatesInLayers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  const int n_hitCandidatesInLayers[VeloUTTracking::n_layers],
  const float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  int bestHitCandidateIndices[VeloUTTracking::n_layers],
  UTHits& ut_hits,
  UTHitOffsets& ut_hit_offsets,
  TrackHelper& helper,
  MiniState& state,
  const float* ut_dxDy,
  const bool forward)
{
  // handle forward / backward cluster search
  int layers[VeloUTTracking::n_layers];
  for ( int i_layer = 0; i_layer < VeloUTTracking::n_layers; ++i_layer ) {
      if ( forward )
        layers[i_layer] = i_layer;
      else
        layers[i_layer] = VeloUTTracking::n_layers - 1 - i_layer;
  }

  // Go through the layers
  bool fourLayerSolution = false;
  int hitCandidateIndices[VeloUTTracking::n_layers];
  for ( int i_hit0 = 0; i_hit0 < n_hitCandidatesInLayers[ layers[0] ]; ++i_hit0 ) {

    const int layer_offset0 = ut_hit_offsets.layer_offset(layers[0]);
    const int hit_index0    = layer_offset0 + hitCandidatesInLayers[ layers[0] ][i_hit0];
    const float xhitLayer0  = x_pos_layers[layers[0]][i_hit0];
    const float zhitLayer0  = ut_hits.zAtYEq0[hit_index0];
    hitCandidateIndices[0] = i_hit0;
    
    for ( int i_hit2 = 0; i_hit2 < n_hitCandidatesInLayers[ layers[2] ]; ++i_hit2 ) {

      const int layer_offset2 = ut_hit_offsets.layer_offset(layers[2]);
      const int hit_index2    = layer_offset2 + hitCandidatesInLayers[ layers[2] ][i_hit2];
      const float xhitLayer2  = x_pos_layers[layers[2]][i_hit2];
      const float zhitLayer2  = ut_hits.zAtYEq0[hit_index2];
      hitCandidateIndices[2] = i_hit2;
      
      const float tx = (xhitLayer2 - xhitLayer0)/(zhitLayer2 - zhitLayer0);
      if( std::abs(tx-state.tx) > PrVeloUTConst::deltaTx2 ) continue;
            
      int IndexBestHit1 = -10;
      float hitTol = PrVeloUTConst::hitTol2;
      for ( int i_hit1 = 0; i_hit1 < n_hitCandidatesInLayers[ layers[1] ]; ++i_hit1 ) {

        const int layer_offset1 = ut_hit_offsets.layer_offset(layers[1]);
        const int hit_index1    = layer_offset1 + hitCandidatesInLayers[ layers[1] ][i_hit1];
        const float xhitLayer1  = x_pos_layers[layers[1]][i_hit1];
        const float zhitLayer1  = ut_hits.zAtYEq0[hit_index1];
       
        const float xextrapLayer1 = xhitLayer0 + tx*(zhitLayer1-zhitLayer0);
        if(std::abs(xhitLayer1 - xextrapLayer1) < hitTol){
          hitTol = std::abs(xhitLayer1 - xextrapLayer1);
          IndexBestHit1 = hit_index1;
          hitCandidateIndices[1] = i_hit1;
        }
      } // loop over layer 1
      
      if( fourLayerSolution && IndexBestHit1 < 0 ) continue;

      int IndexBestHit3 = -10;
      hitTol = PrVeloUTConst::hitTol2;
      for ( int i_hit3 = 0; i_hit3 < n_hitCandidatesInLayers[ layers[3] ]; ++i_hit3 ) {

        const int layer_offset3 = ut_hit_offsets.layer_offset(layers[3]);
        const int hit_index3    = layer_offset3 + hitCandidatesInLayers[ layers[3] ][i_hit3];
        const float xhitLayer3  = x_pos_layers[layers[3]][i_hit3];
        const float zhitLayer3  = ut_hits.zAtYEq0[hit_index3];
        
        const float xextrapLayer3 = xhitLayer2 + tx*(zhitLayer3-zhitLayer2);
        if(std::abs(xhitLayer3 - xextrapLayer3) < hitTol){
          hitTol = std::abs(xhitLayer3 - xextrapLayer3);
          IndexBestHit3 = hit_index3;
          hitCandidateIndices[3] = i_hit3;
        }
      } // loop over layer 3
     
      // -- All hits found
      if ( IndexBestHit1 > 0 && IndexBestHit3 > 0 ) {
        const int hitIndices[4] = {hit_index0, IndexBestHit1, hit_index2, IndexBestHit3};
        simpleFit<4>(x_pos_layers, hitCandidateIndices, bestHitCandidateIndices, hitCandidatesInLayers, ut_hits, hitIndices, helper, state, ut_dxDy);
        
        if(!fourLayerSolution && helper.n_hits > 0){
          fourLayerSolution = true;
        }
        continue;
      }

      // -- Nothing found in layer 3
      if( !fourLayerSolution && IndexBestHit1 > 0 ){
        const int hitIndices[3] = {hit_index0, IndexBestHit1, hit_index2};
        simpleFit<3>(x_pos_layers, hitCandidateIndices, bestHitCandidateIndices, hitCandidatesInLayers, ut_hits, hitIndices, helper, state, ut_dxDy);
        continue;
      }
      // -- Nothing found in layer 1
      if( !fourLayerSolution && IndexBestHit3 > 0 ){
        hitCandidateIndices[1] = hitCandidateIndices[3];  // hit3 saved in second position of hits4fit
        const int hitIndices[3] = {hit_index0, IndexBestHit3, hit_index2};
        simpleFit<3>(x_pos_layers, hitCandidateIndices, bestHitCandidateIndices, hitCandidatesInLayers, ut_hits, hitIndices, helper, state, ut_dxDy);
        continue;
      }
      
    }
  }

  return fourLayerSolution;
}
//=========================================================================
// Create the Velo-UT tracks
//=========================================================================
__host__ __device__ void prepareOutputTrack(
  const Velo::Consolidated::Hits& velo_track_hits,
  const uint velo_track_hit_number,
  const TrackHelper& helper,
  const MiniState& state,
  int hitCandidatesInLayers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  int n_hitCandidatesInLayers[VeloUTTracking::n_layers],
  UTHits& ut_hits,
  UTHitOffsets& ut_hit_offsets,
  const float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  const int hitCandidateIndices[VeloUTTracking::n_layers],
  VeloUTTracking::TrackUT VeloUT_tracks[VeloUTTracking::max_num_tracks],
  int* n_veloUT_tracks,
  const int i_velo_track,
  const float* bdlTable) {

  //== Handle states. copy Velo one, add UT.
  const float zOrigin = (std::fabs(state.ty) > 0.001)
    ? state.z - state.y / state.ty
    : state.z - state.x / state.tx;

  // -- These are calculations, copied and simplified from PrTableForFunction
  const std::array<float,3> var = { state.ty, zOrigin, state.z };

  const int index1 = std::max(0, std::min( 30, int((var[0] + 0.3)/0.6*30) ));
  const int index2 = std::max(0, std::min( 10, int((var[1] + 250)/500*10) ));
  const int index3 = std::max(0, std::min( 10, int( var[2]/800*10)        ));

  assert( masterIndex(index1, index2, index3) < PrUTMagnetTool::N_bdl_vals );
  float bdl = bdlTable[masterIndex(index1, index2, index3)];

  const float bdls[3] = { bdlTable[masterIndex(index1+1, index2,index3)],
                          bdlTable[masterIndex(index1,index2+1,index3)],
                          bdlTable[masterIndex(index1,index2,index3+1)] };
  const float deltaBdl[3]   = { 0.02, 50.0, 80.0 };
  const float boundaries[3] = { -0.3f + float(index1)*deltaBdl[0],
                                -250.0f + float(index2)*deltaBdl[1],
                                0.0f + float(index3)*deltaBdl[2] };

  // -- This is an interpolation, to get a bit more precision
  float addBdlVal = 0.0;
  const float minValsBdl[3] = { -0.3, -250.0, 0.0 };
  const float maxValsBdl[3] = { 0.3, 250.0, 800.0 };
  for(int i=0; i<3; ++i) {
    if( var[i] < minValsBdl[i] || var[i] > maxValsBdl[i] ) continue;
    const float dTab_dVar =  (bdls[i] - bdl) / deltaBdl[i];
    const float dVar = (var[i]-boundaries[i]);
    addBdlVal += dTab_dVar*dVar;
  }
  bdl += addBdlVal;
  // ----

  const float qpxz2p =-1*std::sqrt(1.+state.ty*state.ty)/bdl*3.3356/Gaudi::Units::GeV;
  const float qop = (std::abs(bdl) < 1.e-8) ? 0.0 : helper.bestParams[0]*qpxz2p;

  // -- Don't make tracks that have grossly too low momentum
  // -- Beware of the momentum resolution!
  const float p  = 1.3*std::abs(1/qop);
  const float pt = p*std::sqrt(state.tx*state.tx + state.ty*state.ty);

  if( p < PrVeloUTConst::minMomentum || pt < PrVeloUTConst::minPT ) return;

#ifdef __CUDA_ARCH__
  uint n_tracks = atomicAdd(n_veloUT_tracks, 1);
#else
  (*n_veloUT_tracks)++;
  uint n_tracks = *n_veloUT_tracks - 1;
#endif

  
  const float txUT = helper.bestParams[3];

  // TODO: Maybe have a look and optimize this if possible
  VeloUTTracking::TrackUT track;
  track.hitsNum = 0;
  for (int i=0; i<velo_track_hit_number; ++i) {
    track.addLHCbID(velo_track_hits.LHCbID[i]);
    assert( track.hitsNum < VeloUTTracking::max_track_size);
  }
  track.set_qop( qop );
  track.veloTrackIndex = i_velo_track;
  
  // Adding overlap hits
  for ( int i_hit = 0; i_hit < helper.n_hits; ++i_hit ) {
    const int hit_index = helper.bestHitIndices[i_hit];
    
    track.addLHCbID( ut_hits.LHCbID[hit_index] );
    assert( track.hitsNum < VeloUTTracking::max_track_size);
    
    const int planeCode = ut_hits.planeCode[hit_index];
    const float xhit = x_pos_layers[ planeCode ][ hitCandidateIndices[i_hit] ];
    const float zhit = ut_hits.zAtYEq0[hit_index];

    const int layer_offset = ut_hit_offsets.layer_offset(planeCode);
    for ( int i_ohit = 0; i_ohit < n_hitCandidatesInLayers[planeCode]; ++i_ohit ) {
      const int ohit_index = hitCandidatesInLayers[planeCode][i_ohit];
      const float zohit  = ut_hits.zAtYEq0[layer_offset + ohit_index];
      
      if(zohit==zhit) continue;
      
      const float xohit = x_pos_layers[ planeCode ][ i_ohit];
      const float xextrap = xhit + txUT*(zhit-zohit);
      if( xohit-xextrap < -PrVeloUTConst::overlapTol) continue;
      if( xohit-xextrap > PrVeloUTConst::overlapTol) break;
      
      track.addLHCbID(ut_hits.LHCbID[layer_offset + ohit_index]);
      assert( track.hitsNum < VeloUTTracking::max_track_size);
      
      // -- only one overlap hit
      break;
    }
  }
  assert( n_tracks < VeloUTTracking::max_num_tracks );
  VeloUT_tracks[n_tracks] = track;
}

// ==============================================================================
// -- Finds the hits in a given layer within a certain range
// ==============================================================================
__host__ __device__ void findHits( 
  const uint lowerBoundSectorGroup,
  const uint upperBoundSectorGroup,
  UTHits& ut_hits,
  UTHitOffsets& ut_hit_offsets,
  uint layer_offset,
  const int i_layer,
  const float* ut_dxDy,
  const MiniState& myState, 
  const float xTolNormFact,
  const float invNormFact,
  int hitCandidatesInLayer[VeloUTTracking::max_hit_candidates_per_layer],
  int &n_hitCandidatesInLayer,
  float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer])
{
  const uint posBeg = ut_hit_offsets.sector_group_offset(lowerBoundSectorGroup) - layer_offset;
  const uint posEnd = ut_hit_offsets.sector_group_offset(upperBoundSectorGroup)
    + ut_hit_offsets.sector_group_number_of_hits(upperBoundSectorGroup) - layer_offset;

  const auto zInit = ut_hits.zAtYEq0[layer_offset + posBeg];
  const auto yApprox = myState.y + myState.ty * (zInit - myState.z);
  const auto xOnTrackProto = myState.x + myState.tx*(zInit - myState.z);
  const auto yyProto =       myState.y - myState.ty*myState.z;
  const float dxDy = ut_dxDy[i_layer];

  for (int i=posBeg; i<posEnd; ++i) {
    const auto xx = ut_hits.xAt(layer_offset + i, yApprox, dxDy); 
    const auto dx = xx - xOnTrackProto;
    
    if( dx < -xTolNormFact ) continue;
    if( dx >  xTolNormFact ) continue; 
    
    // -- Now refine the tolerance in Y
    if ( ut_hits.isNotYCompatible( layer_offset + i, yApprox, PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * std::abs(dx*invNormFact)) ) continue;
    
    const auto zz = ut_hits.zAtYEq0[layer_offset + i]; 
    const auto yy = yyProto +  myState.ty*zz;
    const auto xx2 = ut_hits.xAt(layer_offset + i, yy, dxDy);
        
    hitCandidatesInLayer[n_hitCandidatesInLayer] = i;
    x_pos_layers[i_layer][n_hitCandidatesInLayer] = xx2;
    
    n_hitCandidatesInLayer++;

    if ( n_hitCandidatesInLayer >= VeloUTTracking::max_hit_candidates_per_layer )
      printf("%u > %u !! \n", n_hitCandidatesInLayer, VeloUTTracking::max_hit_candidates_per_layer);
    assert( n_hitCandidatesInLayer < VeloUTTracking::max_hit_candidates_per_layer );
  }
  for ( int i_hit = 0; i_hit < n_hitCandidatesInLayer; ++i_hit ) {
    assert( hitCandidatesInLayer[i_hit] < VeloUTTracking::max_numhits_per_event );
  }
}


