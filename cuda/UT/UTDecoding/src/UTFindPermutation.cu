#include "hip/hip_runtime.h"
#include "UTFindPermutation.cuh"
#include "FindPermutation.cuh"
#include <cstdio>

__global__ void ut_find_permutation(
  uint32_t* dev_ut_hits,
  uint32_t* dev_ut_hit_offsets,
  uint* dev_hit_permutations,
  const uint* dev_unique_x_sector_layer_offsets,
  const uint* dev_unique_x_sector_offsets,
  const float* dev_unique_sector_xs)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;
  const uint sector_group_number = blockIdx.y;
  const uint number_of_unique_x_sectors = dev_unique_x_sector_layer_offsets[4];
  
  const UTHitOffsets ut_hit_offsets {dev_ut_hit_offsets, event_number, number_of_unique_x_sectors, dev_unique_x_sector_layer_offsets};
  const UTHits ut_hits {dev_ut_hits, dev_ut_hit_offsets[number_of_events * number_of_unique_x_sectors]};

  // // Prints out all hits
  // if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
  //   printf(" Sector group %i, x %f:\n", sector_group_number, dev_unique_sector_xs[sector_group_number]);
  //   uint group_offset = ut_hit_offsets.sector_group_offset(sector_group_number);
  //   uint n_hits_group = ut_hit_offsets.sector_group_number_of_hits(sector_group_number);
  //   for (int j=0; j<n_hits_group; ++j) {
  //     const auto hit_index = group_offset + j;
  //     // printf("  yBegin = %f, yEnd = %f, zAtYEq0 = %f, xAtYEq0 = %f, weight = %f, highThreshold = %u \n",
  //     //  ut_hits.yBegin[hit_index],
  //     //  ut_hits.yEnd[hit_index],
  //     //  ut_hits.zAtYEq0[hit_index],
  //     //  ut_hits.xAtYEq0[hit_index],
  //     //  ut_hits.weight[hit_index],
  //     //  ut_hits.highThreshold[hit_index]);
  //     printf("  yBegin = %f\n", ut_hits.yBegin[hit_index]);
  //   }
  // }

  const uint sector_group_offset = ut_hit_offsets.sector_group_offset(sector_group_number);
  const uint sector_group_number_of_hits = ut_hit_offsets.sector_group_number_of_hits(sector_group_number);

  if (sector_group_number_of_hits > 0) {
    // Load yBegin into a shared memory container
    // TODO: Find a proper maximum and cover corner cases
    __shared__ float s_y_begin [UTDecoding::ut_max_hits_shared_sector_group];
    assert(sector_group_number_of_hits < UTDecoding::ut_max_hits_shared_sector_group);

    for (int i=threadIdx.x; i<sector_group_number_of_hits; i+=blockDim.x) {
      s_y_begin[i] = ut_hits.yBegin[sector_group_offset + i];
    }

    __syncthreads();

    // Sort according to the natural order in s_y_begin
    // Store the permutation found into dev_hit_permutations
    find_permutation(
      0,
      sector_group_offset,
      sector_group_number_of_hits,
      dev_hit_permutations,
      [] (const int a, const int b) -> int {
        return (s_y_begin[a] > s_y_begin[b]) - (s_y_begin[a] < s_y_begin[b]);
      }
    );
  }
}
