#include "hip/hip_runtime.h"
#include "SearchByTriplet.cuh"
#include "VeloTools.cuh"
#include <cstdio>

/**
 * @brief Fits hits to tracks.
 * 
 * @details In case the tolerances constraints are met,
 *          returns the chi2 weight of the track. Otherwise,
 *          returns FLT_MAX.
 */
__device__ float fit_hit_to_track(
  const Velo::HitBase& h0,
  const Velo::HitBase& h2,
  const float predx,
  const float predy,
  const float scatterDenom2
) {
  // tolerances
  const float x_prediction = h0.x + predx;
  const float dx = fabs(x_prediction - h2.x);
  const bool tolx_condition = dx < VeloTracking::tolerance;

  const float y_prediction = h0.y + predy;
  const float dy = fabs(y_prediction - h2.y);
  const bool toly_condition = dy < VeloTracking::tolerance;

  // Scatter
  const float scatterNum = (dx * dx) + (dy * dy);
  const float scatter = scatterNum * scatterDenom2;

  const bool scatter_condition = scatter < VeloTracking::max_scatter_forwarding;
  const bool condition = tolx_condition && toly_condition && scatter_condition;

  return condition * scatter + !condition * FLT_MAX;
}

/**
 * @brief Performs the track forwarding of forming tracks
 */
__device__ void track_forwarding(
  const float* hit_Xs,
  const float* hit_Ys,
  const float* hit_Zs,
  const float* hit_Phis,
  bool* hit_used,
  uint* tracks_insertPointer,
  uint* ttf_insertPointer,
  uint* weaktracks_insertPointer,
  const Velo::Module* module_data,
  const uint diff_ttf,
  uint* tracks_to_follow,
  Velo::TrackletHits* weak_tracks,
  const uint prev_ttf,
  Velo::TrackletHits* tracklets,
  Velo::TrackHits* tracks,
  const uint number_of_hits
) {
  // Assign a track to follow to each thread
  for (int i=0; i<(diff_ttf + blockDim.x - 1) / blockDim.x; ++i) {
    const uint ttf_element = blockDim.x * i + threadIdx.x;
    if (ttf_element < diff_ttf) {
      const auto fulltrackno = tracks_to_follow[(prev_ttf + ttf_element) % VeloTracking::ttf_modulo];
      const bool track_flag = (fulltrackno & 0x80000000) == 0x80000000;
      const auto skipped_modules = (fulltrackno & 0x70000000) >> 28;
      auto trackno = fulltrackno & 0x0FFFFFFF;
      assert(track_flag ? trackno < VeloTracking::ttf_modulo : trackno < VeloTracking::max_tracks);

      Velo::TrackHits t = track_flag ? Velo::TrackHits{tracklets[trackno]} : tracks[trackno];

      // Load last two hits in h0, h1
      assert(t.hitsNum < VeloTracking::max_track_size);
      const auto h0_num = t.hits[t.hitsNum - 2];
      const auto h1_num = t.hits[t.hitsNum - 1];

      assert(h0_num < number_of_hits);
      const Velo::HitBase h0 {hit_Xs[h0_num], hit_Ys[h0_num], hit_Zs[h0_num]};

      assert(h1_num < number_of_hits);
      const Velo::HitBase h1 {hit_Xs[h1_num], hit_Ys[h1_num], hit_Zs[h1_num]};

      // Track forwarding over t, for all hits in the next module
      // Line calculations
      const auto td = 1.0f / (h1.z - h0.z);
      const auto txn = (h1.x - h0.x);
      const auto tyn = (h1.y - h0.y);
      const auto tx = txn * td;
      const auto ty = tyn * td;
      
      // Find the best candidate
      float best_fit = FLT_MAX;
      unsigned short best_h2;

      // Get candidates by performing a binary search in expected phi
      const auto odd_module_candidates = find_forward_candidates(
        module_data[4],
        tx,
        ty,
        hit_Phis,
        h0,
        [] (const float x, const float y) { return hit_phi_odd(x, y); }
      );

      const auto even_module_candidates = find_forward_candidates(
        module_data[5],
        tx,
        ty,
        hit_Phis,
        h0,
        [] (const float x, const float y) { return hit_phi_even(x, y); }
      );
      
      // Search on both modules in the same for loop
      const int total_odd_candidates = std::get<1>(odd_module_candidates) - std::get<0>(odd_module_candidates);
      const int total_even_candidates = std::get<1>(even_module_candidates) - std::get<0>(even_module_candidates);
      const int total_candidates = total_odd_candidates + total_even_candidates;

      for (int j=0; j<total_candidates; ++j) {
        const int h2_index = j < total_odd_candidates ?
          std::get<0>(odd_module_candidates) + j :
          std::get<0>(even_module_candidates) + j - total_odd_candidates;

        const Velo::HitBase h2 {hit_Xs[h2_index], hit_Ys[h2_index], hit_Zs[h2_index]};

        const auto dz = h2.z - h0.z;
        const auto predx = tx * dz;
        const auto predy = ty * dz;
        const auto scatterDenom2 = 1.f / ((h2.z - h1.z) * (h2.z - h1.z));

        const auto fit = fit_hit_to_track(
          h0,
          h2,
          predx,
          predy,
          scatterDenom2
        );
        
        // We keep the best one found
        if (fit < best_fit) {
          best_fit = fit;
          best_h2 = h2_index;
        }
      }

      // Condition for finding a h2
      if (best_fit != FLT_MAX) {
        // Mark h2 as used
        assert(best_h2 < number_of_hits);
        hit_used[best_h2] = true;

        // Update the tracks to follow, we'll have to follow up
        // this track on the next iteration :)
        assert(t.hitsNum < VeloTracking::max_track_size);
        t.hits[t.hitsNum++] = best_h2;

        // Update the track in the bag
        if (t.hitsNum <= 4) {
          assert(t.hits[0] < number_of_hits);
          assert(t.hits[1] < number_of_hits);
          assert(t.hits[2] < number_of_hits);

          // Also mark the first three as used
          hit_used[t.hits[0]] = true;
          hit_used[t.hits[1]] = true;
          hit_used[t.hits[2]] = true;

          // If it is a track made out of less than or equal than 4 hits,
          // we have to allocate it in the tracks pointer
          trackno = atomicAdd(tracks_insertPointer, 1);
        }

        // Copy the track into tracks
        assert(trackno < VeloTracking::max_tracks);
        tracks[trackno] = t;

        // Add the tracks to the bag of tracks to_follow
        const auto ttfP = atomicAdd(ttf_insertPointer, 1) % VeloTracking::ttf_modulo;
        tracks_to_follow[ttfP] = trackno;
      }
      // A track just skipped a module
      // We keep it for another round
      else if (skipped_modules < VeloTracking::max_skipped_modules) {
        // Form the new mask
        trackno = ((skipped_modules + 1) << 28) | (fulltrackno & 0x8FFFFFFF);

        // Add the tracks to the bag of tracks to_follow
        const auto ttfP = atomicAdd(ttf_insertPointer, 1) % VeloTracking::ttf_modulo;
        tracks_to_follow[ttfP] = trackno;
      }
      // If there are only three hits in this track,
      // mark it as "doubtful"
      else if (t.hitsNum == 3) {
        const auto weakP = atomicAdd(weaktracks_insertPointer, 1) % VeloTracking::ttf_modulo;
        assert(weakP < VeloTracking::max_weak_tracks);
        weak_tracks[weakP] = Velo::TrackletHits{t.hits[0], t.hits[1], t.hits[2]};
      }
      // In the "else" case, we couldn't follow up the track,
      // so we won't be track following it anymore.
    }
  }
}
