#include "SequenceVisitor.cuh"
#include "MuonCatboostEvaluator.cuh"
#include <vector>

template<>
void SequenceVisitor::set_arguments_size<muon_catboost_evaluator_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_muon_catboost_output>(host_buffers.host_number_of_reconstructed_scifi_tracks[0]);
}

template<>
void SequenceVisitor::visit<muon_catboost_evaluator_t>(
  muon_catboost_evaluator_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_reconstructed_scifi_tracks[0]),dim3(32), hip_stream);
  state.set_arguments(
    arguments.offset<dev_muon_catboost_features>(),
    arguments.offset<dev_muon_catboost_output>(),
    constants.dev_muon_catboost_leaf_values,
    constants.dev_muon_catboost_leaf_offsets,
    constants.dev_muon_catboost_split_borders,
    constants.dev_muon_catboost_split_features,
    constants.dev_muon_catboost_tree_depths,
    constants.dev_muon_catboost_tree_offsets,
    constants.muon_catboost_n_trees
  );
  state.invoke();
  
  cudaCheck(hipMemcpyAsync(
    host_buffers.host_muon_catboost_output,
    arguments.offset<dev_muon_catboost_output>(),
    arguments.size<dev_muon_catboost_output>(),
    hipMemcpyDeviceToHost,
    hip_stream
  ));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);
  debug_cout << "IsMuon" << std::endl;
  for(int i = 0; i < host_buffers.host_number_of_reconstructed_scifi_tracks[0]; ++i) {
    debug_cout << host_buffers.host_muon_catboost_output[i] << std::endl;
  }
}
