#include "SequenceVisitor.cuh"
#include "UTPreDecode.cuh"

template<>
void SequenceVisitor::visit<ut_pre_decode_t>(
  ut_pre_decode_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  arguments.set_size<arg::dev_ut_hits>(UTHits::number_of_arrays * host_buffers.host_accumulated_number_of_ut_hits[0]);
  arguments.set_size<arg::dev_ut_hit_count>(runtime_options.number_of_events * constants.host_unique_x_sector_layer_offsets[4]);
  scheduler.setup_next(arguments, sequence_step);

  cudaCheck(hipMemsetAsync(arguments.offset<arg::dev_ut_hit_count>(),
    0,
    arguments.size<arg::dev_ut_hit_count>(),
    hip_stream));

  state.set_opts(dim3(runtime_options.number_of_events), dim3(64, 4), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_ut_raw_input>(),
    arguments.offset<arg::dev_ut_raw_input_offsets>(),
    constants.dev_ut_boards,
    constants.dev_ut_geometry,
    constants.dev_ut_region_offsets,
    constants.dev_unique_x_sector_layer_offsets,
    constants.dev_unique_x_sector_offsets,
    arguments.offset<arg::dev_ut_hit_offsets>(),
    arguments.offset<arg::dev_ut_hits>(),
    arguments.offset<arg::dev_ut_hit_count>()
  );

  state.invoke();
}
