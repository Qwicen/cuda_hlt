#include "SequenceVisitor.cuh"
#include "UTCalculateNumberOfHits.cuh"

template<>
void SequenceVisitor::visit<ut_calculate_number_of_hits_t>(
  ut_calculate_number_of_hits_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Set arguments and reserve memory
  arguments.set_size<arg::dev_ut_raw_input>(runtime_options.host_ut_events_size);
  arguments.set_size<arg::dev_ut_raw_input_offsets>(runtime_options.host_ut_event_offsets_size);
  arguments.set_size<arg::dev_ut_hit_offsets>(runtime_options.number_of_events * constants.host_unique_x_sector_layer_offsets[4] + 1);
  scheduler.setup_next(arguments, sequence_step);

  // Setup opts and arguments for kernel call
  cudaCheck(hipMemcpyAsync(arguments.offset<arg::dev_ut_raw_input>(),
    runtime_options.host_ut_events,
    runtime_options.host_ut_events_size,
    hipMemcpyHostToDevice,
    hip_stream));

  cudaCheck(hipMemcpyAsync(arguments.offset<arg::dev_ut_raw_input_offsets>(),
    runtime_options.host_ut_event_offsets,
    runtime_options.host_ut_event_offsets_size * sizeof(uint32_t),
    hipMemcpyHostToDevice,
    hip_stream));

  cudaCheck(hipMemsetAsync(arguments.offset<arg::dev_ut_hit_offsets>(),
    0,
    arguments.size<arg::dev_ut_hit_offsets>(),
    hip_stream));

  state.set_opts(dim3(runtime_options.number_of_events), dim3(64, 4), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_ut_raw_input>(),
    arguments.offset<arg::dev_ut_raw_input_offsets>(),
    constants.dev_ut_boards,
    constants.dev_ut_region_offsets,
    constants.dev_unique_x_sector_layer_offsets,
    constants.dev_unique_x_sector_offsets,
    arguments.offset<arg::dev_ut_hit_offsets>()
  );

  // Invoke kernel
  state.invoke();

  // // Print UT hit count per event per layer
  // std::vector<uint> host_ut_hit_count (number_of_events * constants.host_unique_x_sector_layer_offsets[4] + 1);
  // cudaCheck(hipMemcpyAsync(host_ut_hit_count.data(), arguments.offset<arg::dev_ut_hit_offsets>(), argen.size<arg::dev_ut_hit_offsets>(number_of_events * constants.host_unique_x_sector_layer_offsets[4] + 1), hipMemcpyDeviceToHost, stream));
  // hipEventRecord(cuda_generic_event, stream);
  // hipEventSynchronize(cuda_generic_event);
  // for (int e=0; e<number_of_events; ++e) {
  //   info_cout << "Event " << e << ", #hits per layer: ";
  //   uint32_t* count = host_ut_hit_count.data() + e * constants.host_unique_x_sector_layer_offsets[4];
  //   for (uint32_t i = 0; i < constants.host_unique_x_sector_layer_offsets[4]; ++i) {
  //     info_cout << count[i] << ", ";
  //   }
  //   info_cout << std::endl;
  // }
}
