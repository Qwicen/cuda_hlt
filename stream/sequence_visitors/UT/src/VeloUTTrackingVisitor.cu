#include "SequenceVisitor.cuh"
#include "VeloUT.cuh"

template<>
void SequenceVisitor::visit<veloUT_t>(
  veloUT_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  arguments.set_size<arg::dev_veloUT_tracks>(runtime_options.number_of_events * VeloUTTracking::max_num_tracks);
  arguments.set_size<arg::dev_atomics_veloUT>(runtime_options.number_of_events * VeloUTTracking::num_atomics);
  scheduler.setup_next(arguments, sequence_step);

  state.set_opts(dim3(runtime_options.number_of_events), dim3(32), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_ut_hits>(),
    arguments.offset<arg::dev_ut_hit_offsets>(),
    arguments.offset<arg::dev_atomics_storage>(),
    arguments.offset<arg::dev_velo_track_hit_number>(),
    arguments.offset<arg::dev_velo_track_hits>(),
    arguments.offset<arg::dev_velo_states>(),
    arguments.offset<arg::dev_veloUT_tracks>(),
    arguments.offset<arg::dev_atomics_veloUT>(),
    constants.dev_ut_magnet_tool,
    constants.dev_ut_dxDy,
    constants.dev_unique_x_sector_layer_offsets,
    constants.dev_unique_x_sector_offsets,
    constants.dev_unique_sector_xs
  );

  state.invoke();

  // TODO: Maybe this should not go here
  // Fetch all UT tracks
  cudaCheck(hipMemcpyAsync(host_buffers.host_atomics_veloUT,
    arguments.offset<arg::dev_atomics_veloUT>(),
    arguments.size<arg::dev_atomics_veloUT>(),
    hipMemcpyDeviceToHost,
    hip_stream));

  cudaCheck(hipMemcpyAsync(host_buffers.host_veloUT_tracks,
    arguments.offset<arg::dev_veloUT_tracks>(),
    arguments.size<arg::dev_veloUT_tracks>(),
    hipMemcpyDeviceToHost, 
    hip_stream));
}
