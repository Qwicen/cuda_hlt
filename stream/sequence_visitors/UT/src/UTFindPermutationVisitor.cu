#include "SequenceVisitor.cuh"
#include "UTFindPermutation.cuh"

template<>
void SequenceVisitor::visit<ut_find_permutation_t>(
  ut_find_permutation_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  arguments.set_size<arg::dev_ut_hit_permutations>(host_buffers.host_accumulated_number_of_ut_hits[0]);
  scheduler.setup_next(arguments, sequence_step);

  state.set_opts(dim3(runtime_options.number_of_events, constants.host_unique_x_sector_layer_offsets[4]), dim3(16), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_ut_hits>(),
    arguments.offset<arg::dev_ut_hit_offsets>(),
    arguments.offset<arg::dev_ut_hit_permutations>(),
    constants.dev_unique_x_sector_layer_offsets,
    constants.dev_unique_x_sector_offsets,
    constants.dev_unique_sector_xs
  );

  state.invoke();
}
