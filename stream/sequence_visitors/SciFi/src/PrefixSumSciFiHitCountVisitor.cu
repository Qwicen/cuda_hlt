#include "SequenceVisitor.cuh"
#include "PrefixSum.cuh"

template<>
void SequenceVisitor::visit<prefix_sum_reduce_scifi_hits_t>(
  prefix_sum_reduce_scifi_hits_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Prefix sum: Reduce
  const uint total_number_of_zones = runtime_options.number_of_events * SciFi::Constants::n_zones;
  const size_t prefix_sum_auxiliary_array_size = (total_number_of_zones + 511) / 512;
  arguments.set_size<arg::dev_prefix_sum_auxiliary_array_4>(prefix_sum_auxiliary_array_size);
  scheduler.setup_next(arguments, sequence_step);

  state.set_opts(dim3(prefix_sum_auxiliary_array_size), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_scifi_hit_count>(),
    arguments.offset<arg::dev_prefix_sum_auxiliary_array_4>(),
    total_number_of_zones
  );

  state.invoke();
}

template<>
void SequenceVisitor::visit<prefix_sum_single_block_scifi_hits_t>(
  prefix_sum_single_block_scifi_hits_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Prefix sum: Single block
  scheduler.setup_next(arguments, sequence_step);

  const uint total_number_of_zones = runtime_options.number_of_events * SciFi::Constants::n_zones;
  const size_t prefix_sum_auxiliary_array_size = (total_number_of_zones + 511) / 512;
  state.set_opts(dim3(1), dim3(1024), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_scifi_hit_count>() + total_number_of_zones,
    arguments.offset<arg::dev_prefix_sum_auxiliary_array_4>(),
    prefix_sum_auxiliary_array_size
  );

  state.invoke();
}

template<>
void SequenceVisitor::visit<prefix_sum_scan_scifi_hits_t>(
  prefix_sum_scan_scifi_hits_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Prefix sum: Scan
  scheduler.setup_next(arguments, sequence_step);

  const uint total_number_of_zones = runtime_options.number_of_events * SciFi::Constants::n_zones;
  const size_t prefix_sum_auxiliary_array_size = (total_number_of_zones + 511) / 512;
  const uint pss_scifi_hits_blocks = prefix_sum_auxiliary_array_size==1 ? 1 : (prefix_sum_auxiliary_array_size-1);
  state.set_opts(dim3(pss_scifi_hits_blocks), dim3(512), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_scifi_hit_count>(),
    arguments.offset<arg::dev_prefix_sum_auxiliary_array_4>(),
    total_number_of_zones
  );

  state.invoke();

  // Fetch total number of hits
  cudaCheck(hipMemcpyAsync(host_buffers.host_accumulated_number_of_scifi_hits,
    arguments.offset<arg::dev_scifi_hit_count>() + total_number_of_zones,
    sizeof(uint),
    hipMemcpyDeviceToHost,
    hip_stream));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  // info_cout << "Total SciFi cluster estimate: " << *host_accumulated_number_of_scifi_hits << std::endl;
}
