#include "SequenceVisitor.cuh"
#include "EstimateInputSize.cuh"

template<>
void SequenceVisitor::visit<estimate_input_size_t>(
  estimate_input_size_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Set arguments and reserve memory
  arguments.set_size<arg::dev_raw_input>(runtime_options.host_velopix_events_size);
  arguments.set_size<arg::dev_raw_input_offsets>(runtime_options.host_velopix_event_offsets_size);
  arguments.set_size<arg::dev_estimated_input_size>(runtime_options.number_of_events * VeloTracking::n_modules + 1);
  arguments.set_size<arg::dev_module_cluster_num>(runtime_options.number_of_events * VeloTracking::n_modules);
  arguments.set_size<arg::dev_module_candidate_num>(runtime_options.number_of_events);
  arguments.set_size<arg::dev_cluster_candidates>(runtime_options.number_of_events * VeloClustering::max_candidates_event);
  scheduler.setup_next(arguments, sequence_step);

  // Setup opts and arguments for kernel call
  state.set_opts(dim3(runtime_options.number_of_events), dim3(32, 26), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_raw_input>(),
    arguments.offset<arg::dev_raw_input_offsets>(),
    arguments.offset<arg::dev_estimated_input_size>(),
    arguments.offset<arg::dev_module_cluster_num>(),
    arguments.offset<arg::dev_module_candidate_num>(),
    arguments.offset<arg::dev_cluster_candidates>(),
    constants.dev_velo_candidate_ks
  );

  // Fetch required arguments
  cudaCheck(hipMemcpyAsync(arguments.offset<arg::dev_raw_input>(), runtime_options.host_velopix_events, arguments.size<arg::dev_raw_input>(), hipMemcpyHostToDevice, hip_stream));
  cudaCheck(hipMemcpyAsync(arguments.offset<arg::dev_raw_input_offsets>(), runtime_options.host_velopix_event_offsets, arguments.size<arg::dev_raw_input_offsets>(), hipMemcpyHostToDevice, hip_stream));
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  // Kernel call
  state.invoke();
}
