#include "SequenceVisitor.cuh"
#include "PrefixSum.cuh"

template<>
void SequenceVisitor::visit<prefix_sum_reduce_velo_clusters_t>(
  prefix_sum_reduce_velo_clusters_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Convert the estimated sizes to module hit start format (argument_offsets)
  // Set arguments and reserve memory
  arguments.set_size<arg::dev_cluster_offset>(runtime_options.number_of_events);
  scheduler.setup_next(arguments, sequence_step);
  
  // Setup sequence step
  const auto prefix_sum_blocks = (VeloTracking::n_modules * runtime_options.number_of_events + 511) / 512;
  state.set_opts(dim3(prefix_sum_blocks), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_estimated_input_size>(),
    arguments.offset<arg::dev_cluster_offset>(),
    VeloTracking::n_modules * runtime_options.number_of_events
  );

  // Kernel call
  state.invoke();
}

template<>
void SequenceVisitor::visit<prefix_sum_single_block_velo_clusters_t>(
  prefix_sum_single_block_velo_clusters_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // TODO: Make prefix sum use less repeated code
  const auto prefix_sum_blocks = (VeloTracking::n_modules * runtime_options.number_of_events + 511) / 512;

  // Prefix Sum Single Block
  scheduler.setup_next(arguments, sequence_step);
  state.set_opts(dim3(1), dim3(1024), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_estimated_input_size>() + VeloTracking::n_modules * runtime_options.number_of_events,
    arguments.offset<arg::dev_cluster_offset>(),
    prefix_sum_blocks
  );

  state.invoke();
}

template<>
void SequenceVisitor::visit<prefix_sum_scan_velo_clusters_t>(
  prefix_sum_scan_velo_clusters_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Prefix sum scan
  scheduler.setup_next(arguments, sequence_step);

  const auto prefix_sum_blocks = (VeloTracking::n_modules * runtime_options.number_of_events + 511) / 512;
  const auto prefix_sum_scan_blocks = prefix_sum_blocks==1 ? 1 : (prefix_sum_blocks-1);
  state.set_opts(dim3(prefix_sum_scan_blocks), dim3(512), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_estimated_input_size>(),
    arguments.offset<arg::dev_cluster_offset>(),
    VeloTracking::n_modules * runtime_options.number_of_events
  );
  state.invoke();

  // Fetch the number of hits we require
  cudaCheck(hipMemcpyAsync(host_buffers.host_total_number_of_velo_clusters,
    arguments.offset<arg::dev_estimated_input_size>() + runtime_options.number_of_events * VeloTracking::n_modules,
    sizeof(uint),
    hipMemcpyDeviceToHost,
    hip_stream));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);
}
