#include "SequenceVisitor.cuh"
#include "WeakTracksAdder.cuh"

template<>
void SequenceVisitor::visit<weak_tracks_adder_t>(
  weak_tracks_adder_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  scheduler.setup_next(arguments, sequence_step);

  // Setup opts and arguments
  state.set_opts(dim3(runtime_options.number_of_events), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_velo_cluster_container>(),
    arguments.offset<arg::dev_estimated_input_size>(),
    arguments.offset<arg::dev_tracks>(),
    arguments.offset<arg::dev_weak_tracks>(),
    arguments.offset<arg::dev_hit_used>(),
    arguments.offset<arg::dev_atomics_storage>()
  );
  
  state.invoke();
}
